#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include "header.h"

__device__ void generateNormal(int n, float * dev,hiprandState_t *st)	   
{
	const int blockId = blockIdx.x+blockIdx.y*gridDim.x;
	const int threadId = (blockId*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x)+threadIdx.x);
	hiprand_init(n,blockIdx.x,0,st);
	for (int i=0; i<n;i++)
	{
		
          dev[i] = hiprand_normal(&st[blockIdx.x]);
	  //printf("val %f\n",dev[i]);
	}
}

__device__ float endOptionValue(float S,float K,
	   	 		float i, float mu,
				float v, float time,float type)
{

	float optVal, result;
	optVal = S * __expf(mu*time+v*sqrt(time)*i);

	if ( type == 1.0)
 	result = optVal -K;

	if (type == 2.0)
 	result = K - optVal;
	return (result > 0.0) ? result :0.0;
}       	      


__global__ void __MonteCarlo__ (float*d_price,float *d_strike, float*d_type, int n_opt,float*dev,hiprandState_t *states,float*temp)
{	const int numBlocks = gridDim.x*gridDim.y;
	const int thrPer = blockDim.x * blockDim.y;
	const int steps=365;
	float sum;
	const int blockId = blockIdx.x+blockIdx.y*gridDim.x;
	const int threadId = (blockId*(blockDim.x*blockDim.y)+(threadIdx.y*blockDim.x)+threadIdx.x);
	
	//printf("value threadId %d\n",threadId);	
	
	generateNormal(steps,temp,&states[threadId]);
	__syncthreads();

	for (int i =0;i<n_opt;i++)
	{   sum=0.0f;
	    //printf("value of %d, %f\n",threadId,temp[365]);
	    //{
		     for (int j=0;j<steps;j++)
		    { 
		
			//printf("val  %f\n",temp[j]);
		     	sum+=endOptionValue(100.0,d_strike[i],(blockIdx.x*temp[j]),0.1f,0.2f,1.0,d_type[i]);
		    
		     }
		/* average solution */
		d_price[blockId]= sum/(thrPer*gridDim.x);
		
	}
}

extern "C" void launchKernel(float*price,float *strike, float*type, int n_opt)
{
	int numBlocks =0;
	int thrPer =0;
	const int blocksPerOption = (n_opt < 16)? 64 : 16;
	const int thr = 16;
	hiprandState_t *states;
	float *d_price,*d_strike,*d_type,*dev,*temp;
	hipMalloc((void**)&d_price, n_opt*sizeof(float));
	hipMalloc((void**)&d_strike,n_opt*sizeof(float));
	hipMalloc((void**)&d_type,n_opt*sizeof(float));
	hipMalloc((void**)&dev,365*sizeof(float));
	hipMalloc((void**)&temp,365*sizeof(float));
	const int doMultiBlock = (blocksPerOption *thrPer/n_opt)>=8192;
	
	//hipEvent_t start, stop;
	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	/* we use multiple blocks to price one option */
	if ( doMultiBlock )
	{
		/*2D grid */
		dim3 grid(blocksPerOption,n_opt);
		dim3 block(thr, thr);
		numBlocks = n_opt*blocksPerOption;
		thrPer = thr*thr;
	}

	
        else 
	{
		/* we use one block per option */
       		dim3 blocks(n_opt);
		dim3 threads(thr*thr);
		numBlocks = n_opt;
		thrPer = thr*thr;
	}

	hipMalloc((void**)&states,numBlocks*thrPer*365*sizeof(hiprandState_t));
	

	hipMemcpy(d_strike,strike,n_opt*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_type,type,n_opt*sizeof(float),hipMemcpyHostToDevice);
	//hipEventRecord(start,0);			
	__MonteCarlo__<<<1024,256>>>(d_price,strike,d_type,n_opt,dev,states,temp);
	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
	//float Time;
	//hipEventElapsedTime(&Time,start,stop);
	//printf("Time to price options %f ms\n",Time);

	/* copy prices of options to CPU/Host*/
	hipMemcpy(price,d_price, n_opt*sizeof(float),hipMemcpyDeviceToHost);
	hipFree(states);
	hipFree(dev);
	hipFree(d_type);
	hipFree(d_strike);
	hipFree(d_price);
	hipFree(temp);
}

